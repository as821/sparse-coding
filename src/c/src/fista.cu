#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <execinfo.h>
#include <sys/time.h>
#include <stdbool.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>


void print_stack_trace();
#define CHECK(x)                                                                                    \
{                                                                                                   \
    if(!(x)) {                                                                                      \
        printf("ERROR (line %d, file:%s) (%d): %s\n", __LINE__, __FILE__, errno, strerror(errno));  \
        print_stack_trace();                                                                        \
        exit(EXIT_FAILURE);                                                                         \
    }                                                                                               \
}

#define CHECK_CUDA_NORET(func)                                                  \
{                                                                               \
    hipError_t status = (func);                                                \
    if (status != hipSuccess) {                                                \
        printf("CUDA API failed at line %d with error: %s (%d) (%s)\n",         \
               __LINE__, hipGetErrorString(status), status, __FILE__);         \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define CHECK_CUBLAS_NORET(func)                                                \
{                                                                               \
    hipblasStatus_t status = (func);                                             \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                      \
        printf("CUBLAS API failed at line %d with error: (%d) (%s)\n",          \
               __LINE__, status, __FILE__);                                     \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

__global__ void y_update(size_t n, float* Y, float* z_prev, float alpha_L, float mlt, float* diff_norm, float* prev_z_norm) {
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;
    
    float thread_local_diff_norm = 0;
    float thread_local_prev_z_norm = 0;
    
    // NOTE(as): 2 read + 2 writes per iteration (assuming thread local var are cached properly)
    for(int idx = index; idx < n; idx += stride) {
        float Y_val = Y[idx] < alpha_L ? 0.0f : Y[idx] - alpha_L;

        float Y_prev = z_prev[idx];
        z_prev[idx] = Y_val;
        thread_local_prev_z_norm += Y_prev * Y_prev;

        float diff = Y_val - Y_prev;
        thread_local_diff_norm += diff * diff;
        
        Y_val += mlt * diff;
        Y[idx] = Y_val;
    }

    extern __shared__ float shmem[];
    float* shared_diff_norm = shmem;
    float* shared_prev_z_norm = &shmem[blockDim.x];

    // tree-based reduction of thread-local norm values for all threads in the block
    shared_diff_norm[tid] = thread_local_diff_norm;
    shared_prev_z_norm[tid] = thread_local_prev_z_norm;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (tid < s) {
            shared_diff_norm[tid] += shared_diff_norm[tid + s];
            shared_prev_z_norm[tid] += shared_prev_z_norm[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(diff_norm, shared_diff_norm[0]);
        atomicAdd(prev_z_norm, shared_prev_z_norm[0]);
    }
}

void print_norm_host(float* arr, size_t sz, const char* str) {
    double norm = 0;
    for(int idx = 0; idx < sz; idx++) {
        norm += (double)arr[idx] * (double)arr[idx];
    }
    norm = sqrt(norm);
    printf("%s: %f\n", str, norm);
}

void print_norm(float* arr_dev, size_t sz, const char* str) {
    float* arr = (float*)malloc(sz * sizeof(float));
    CHECK(arr);
    CHECK_CUDA_NORET(hipMemcpy((void*)arr, arr_dev, sz * sizeof(float), hipMemcpyDeviceToHost))
    print_norm_host(arr, sz, str);
    free(arr);
}

void cuda_log_time_diff(char* msg, hipEvent_t* start, hipEvent_t* stop) {
    float milli = 0;
    hipEventElapsedTime(&milli, *start, *stop);
    milli /= 1000;      // ms -> s
    printf("%s: %f\n", msg, milli);
}

void log_time_diff(char* msg, struct timeval* start, struct timeval* stop) {
    double start_ms = (((double)start->tv_sec)*1000)+(((double)start->tv_usec)/1000);
    double stop_ms = (((double)stop->tv_sec)*1000)+(((double)stop->tv_usec)/1000);
    double diff_in_sec = (stop_ms - start_ms)/1000;
    printf("%s: %f\n", msg, diff_in_sec);
}


extern "C" {
int fista(float* __restrict__ X_host, float* __restrict__ basis_host, float* __restrict__ Z_host, int n_samples, int inp_dim, int dict_sz, float lr, float alpha_L, int n_iter, float converge_thresh) {
    CHECK(X_host);
    CHECK(basis_host);
    CHECK(Z_host);

    struct timeval actual_start, handle_time, init, exec;
    gettimeofday(&actual_start, NULL);


    // X: n_samples x inp_dim
    // basis: inp_dim x dict_sz
    // Z: n_samples x dict_sz

    // TODO(as): (very) slow the first time it is called in a process...
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    gettimeofday(&handle_time, NULL);


    // TODO(as): bunch of faster + less precise BLAS options here https://docs.nvidia.com/cuda/cublas/#cublasoperation-t
    // TODO(as): CUTLASS? https://github.com/NVIDIA/cutlass/blob/main/examples/45_dual_gemm/dual_gemm.cu
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F_PEDANTIC;


    // TODO(as): page locking + async transfers. ways to move these calls outside of this function so not called on each iteration?
    float *X, *basis;
    size_t x_n_el = n_samples * inp_dim;
    size_t x_sz = x_n_el * sizeof(float);
    size_t basis_sz = inp_dim * dict_sz * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&X, x_sz))
    CHECK_CUDA_NORET(hipMalloc((void**)&basis, basis_sz))
    CHECK_CUDA_NORET(hipMemcpy((void*)X, X_host, x_sz, hipMemcpyHostToDevice))
    CHECK_CUDA_NORET(hipMemcpy((void*)basis, basis_host, basis_sz, hipMemcpyHostToDevice))

    float *residual, *z_prev, *Y;
    size_t z_n_el = dict_sz * n_samples;
    size_t z_sz = z_n_el * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&residual, x_sz))
    CHECK_CUDA_NORET(hipMalloc((void**)&z_prev, z_sz))
    CHECK_CUDA_NORET(hipMalloc((void**)&Y, z_sz))
    CHECK_CUDA_NORET(hipMemset(z_prev, 0, z_sz))
    CHECK_CUDA_NORET(hipMemset(Y, 0, z_sz))

    float norms_host[2];
    float* norms;
    size_t norm_sz = 2 * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&norms, norm_sz))

    gettimeofday(&init, NULL);

    float tk = 1, tk_prev = 1;
    int itr;
    for(itr = 0; itr < n_iter; itr++) {

        hipEvent_t start, blas, k_start, k_exec, k_end;
        hipEventCreate(&start);
        hipEventCreate(&blas);
        hipEventCreate(&k_start);
        hipEventCreate(&k_exec);
        hipEventCreate(&k_end);
        hipEventRecord(start);

        // residual = x - (z @ basis.T)
        CHECK_CUDA_NORET(hipMemcpy((void*)residual, X, x_sz, hipMemcpyDeviceToDevice))
        {
            // cublas assumes column-major but we have row major
            // https://i.sstatic.net/IvZPe.png
            float alpha = -1.0f;
            float beta = 1.0f;
            CHECK_CUBLAS_NORET(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inp_dim, n_samples, dict_sz, &alpha, basis, HIP_R_32F, dict_sz, Y, HIP_R_32F, dict_sz, &beta, residual, HIP_R_32F, inp_dim, compute_type, HIPBLAS_GEMM_DEFAULT));    
        }

        // mm = residual @ basis
        // z += lr * mm
        {
            // cublas assumes column-major but we have row major
            // https://i.sstatic.net/IvZPe.png
            float beta = 1.0f;
            CHECK_CUBLAS_NORET(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dict_sz, n_samples, inp_dim, &lr, basis, HIP_R_32F, dict_sz, residual, HIP_R_32F, inp_dim, &beta, Y, HIP_R_32F, dict_sz, compute_type, HIPBLAS_GEMM_DEFAULT));
        }

        hipEventRecord(blas);

        // Y-update multiplier
        tk_prev = tk;
        tk = (1 + sqrtf(1 + 4 * tk * tk)) / 2;
        float mlt = (tk_prev - 1) / tk;
        CHECK_CUDA_NORET(hipMemset(norms, 0, norm_sz))
        hipEventRecord(k_start);
        
        int block_sz = 256;
        int n_blocks = (z_n_el + block_sz - 1) / block_sz;       // ceil(z_n_el / block_sz)
        int smem_sz = 2 * block_sz * sizeof(float);
        y_update<<<n_blocks, block_sz, smem_sz>>>(z_n_el, Y, z_prev, alpha_L, mlt, norms, &norms[1]);
        hipEventRecord(k_exec);

        CHECK_CUDA_NORET(hipMemcpy((void*)norms_host, norms, norm_sz, hipMemcpyDeviceToHost))

        // Frobenius norm can be defined as the L2 norm of the flattened matrix
        float diff_norm = norms_host[0];
        float prev_z_norm = norms_host[1];
        float norm_ratio = diff_norm / prev_z_norm;
        norm_ratio = sqrtf(norm_ratio);         // equivalent to sqrtf(diff_norm) / sqrtf(prev_z_norm)
        hipEventRecord(k_end);


        // printf("%d: %f %f\n", itr, sqrtf(diff_norm), sqrtf(prev_z_norm));
        printf("\33[2K\r%d / %d", itr, n_iter);
        fflush(stdout);


        hipEventSynchronize(k_end);
        cuda_log_time_diff("\n\tblas", &start, &blas);
        cuda_log_time_diff("\tk_start", &blas, &k_start);
        cuda_log_time_diff("\tk_exec", &k_start, &k_exec);
        cuda_log_time_diff("\tk_end", &k_exec, &k_end);
        float milli = 0;
        hipEventElapsedTime(&milli, k_start, k_exec);
        printf("\tbandwidth: %f (GB/s)\n", z_sz * 4 / milli / 1e6);     // 2 read + 2 write per iteration   

        if(itr != 0 && norm_ratio < converge_thresh)
            break;
    }
    printf("\n");

    // memcpy(Z, z_prev, dict_sz * n_samples * sizeof(float));
    CHECK_CUDA_NORET(hipMemcpy((void*)Z_host, z_prev, z_sz, hipMemcpyDeviceToHost))


    CHECK_CUDA_NORET(hipFree(residual))
    CHECK_CUDA_NORET(hipFree(z_prev))
    CHECK_CUDA_NORET(hipFree(norms))
    CHECK_CUDA_NORET(hipFree(Y))
    CHECK_CUDA_NORET(hipFree(X))
    CHECK_CUDA_NORET(hipFree(basis))

    hipblasDestroy(handle);


    gettimeofday(&exec, NULL);


    log_time_diff("\n\n\thandle", &actual_start, &handle_time);
    log_time_diff("\tinit", &handle_time, &init);
    log_time_diff("\texec", &init, &exec);
    return itr;
}
}


void print_stack_trace() {
    void *array[100];
    size_t size;
    char **strings;
    size_t i;

    size = backtrace(array, 100);
    strings = backtrace_symbols(array, size);

    printf("Stack trace:\n");
    for (i = 0; i < size; i++) {
        printf("\t%s\n", strings[i]);
    }

    free(strings);
}


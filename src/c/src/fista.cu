#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <execinfo.h>
#include <sys/time.h>
#include <stdbool.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>


#define DEBUG false

// https://www.ceremade.dauphine.fr/~carlier/FISTA
// http://robotics.stanford.edu/~ang/papers/icml09-LargeScaleUnsupervisedDeepLearningGPU.pdf
// https://proceedings.neurips.cc/paper_files/paper/2006/file/2d71b2ae158c7c5912cc0bbde2bb9d95-Paper.pdf
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/3_CUDA_Features/cudaCompressibleMemory/compMalloc.cpp

void print_stack_trace();
#define CHECK(x)                                                                                    \
{                                                                                                   \
    if(!(x)) {                                                                                      \
        printf("ERROR (line %d, file:%s) (%d): %s\n", __LINE__, __FILE__, errno, strerror(errno));  \
        print_stack_trace();                                                                        \
        exit(EXIT_FAILURE);                                                                         \
    }                                                                                               \
}

#define CHECK_CUDA_NORET(func)                                                  \
{                                                                               \
    hipError_t status = (func);                                                \
    if (status != hipSuccess) {                                                \
        printf("CUDA API failed at line %d with error: %s (%d) (%s)\n",         \
               __LINE__, hipGetErrorString(status), status, __FILE__);         \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define CHECK_CUBLAS_NORET(func)                                                \
{                                                                               \
    hipblasStatus_t status = (func);                                             \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                      \
        printf("CUBLAS API failed at line %d with error: (%d) (%s)\n",          \
               __LINE__, status, __FILE__);                                     \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}


hipError_t setProp(hipMemAllocationProp *prop, bool UseCompressibleMemory)
{
    hipDevice_t currentDevice;
    if (hipCtxGetDevice(&currentDevice) != hipSuccess)
        return hipErrorOutOfMemory;

    memset(prop, 0, sizeof(hipMemAllocationProp));
    prop->type = hipMemAllocationTypePinned;
    prop->location.type = hipMemLocationTypeDevice;
    prop->location.id = currentDevice;

    if (UseCompressibleMemory)
        prop->allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

    return hipSuccess;
}

hipError_t allocateCompressible(void **adr, size_t size)
{
    bool UseCompressibleMemory = true;
    hipMemAllocationProp prop = {};
    hipError_t err = setProp(&prop, UseCompressibleMemory);
    if (err != hipSuccess)
        return err;

    size_t granularity = 0;
    if (hipMemGetAllocationGranularity(&granularity, &prop,
                                      hipMemAllocationGranularityMinimum) != hipSuccess)
        return hipErrorOutOfMemory;
    size = ((size - 1) / granularity + 1) * granularity;
    hipDeviceptr_t dptr;
    if (hipMemAddressReserve(&dptr, size, 0, 0, 0) != hipSuccess)
        return hipErrorOutOfMemory;

    hipMemGenericAllocationHandle_t allocationHandle;
    if (hipMemCreate(&allocationHandle, size, &prop, 0) != hipSuccess)
        return hipErrorOutOfMemory;

    // Check if cuMemCreate was able to allocate compressible memory.
    if (UseCompressibleMemory) {
        hipMemAllocationProp allocationProp = {};
        hipMemGetAllocationPropertiesFromHandle(&allocationProp, allocationHandle);
        if (allocationProp.allocFlags.compressionType != CU_MEM_ALLOCATION_COMP_GENERIC) {
            printf("Could not allocate compressible memory... so waiving execution\n");
            CHECK(false);
        }
    }

    if (hipMemMap(dptr, size, 0, allocationHandle, 0) != hipSuccess)
        return hipErrorOutOfMemory;

    if (hipMemRelease(allocationHandle) != hipSuccess)
        return hipErrorOutOfMemory;

    hipMemAccessDesc accessDescriptor;
    accessDescriptor.location.id = prop.location.id;
    accessDescriptor.location.type = prop.location.type;
    accessDescriptor.flags = hipMemAccessFlagsProtReadWrite;

    if (hipMemSetAccess(dptr, size, &accessDescriptor, 1) != hipSuccess)
        return hipErrorOutOfMemory;

    *adr = (void *)dptr;
    return hipSuccess;
}

hipError_t freeCompressible(void *ptr, size_t size)
{
    bool UseCompressibleMemory = true;
    hipMemAllocationProp prop = {};
    hipError_t err = setProp(&prop, UseCompressibleMemory);
    if (err != hipSuccess)
        return err;

    size_t granularity = 0;
    if (hipMemGetAllocationGranularity(&granularity, &prop,
                                      hipMemAllocationGranularityMinimum) != hipSuccess)
        return hipErrorOutOfMemory;
    size = ((size - 1) / granularity + 1) * granularity;

    if (ptr == NULL)
        return hipSuccess;
    if (hipMemUnmap((hipDeviceptr_t)ptr, size) != hipSuccess ||
        hipMemAddressFree((hipDeviceptr_t)ptr, size) != hipSuccess)
        return hipErrorInvalidValue;
    return hipSuccess;
}


__device__ __forceinline__ float branchless_relu(float x) {
    return x * (x > 0.0f);
}

template <unsigned int block_sz>
__device__ __forceinline__ void warp_reduce(volatile float* sdata, int tid) {
    if(block_sz >= 64)
        sdata[tid] += sdata[tid + 32];
    if(block_sz >= 32)
        sdata[tid] += sdata[tid + 16];
    if(block_sz >= 16)
        sdata[tid] += sdata[tid + 8];
    if(block_sz >= 8)
        sdata[tid] += sdata[tid + 4];
    if(block_sz >= 4)
        sdata[tid] += sdata[tid + 2];
    if(block_sz >= 2)
        sdata[tid] += sdata[tid + 1];
}

template <unsigned int block_sz, unsigned int n_el_per_thread>
__global__ void y_update(size_t n, float4* __restrict__ Y, float4* __restrict__ z_prev, float alpha_L, float mlt, float* __restrict__ diff_norm, float* __restrict__ prev_z_norm) {
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;
    
    float thread_local_diff_norm = 0;
    float thread_local_prev_z_norm = 0;

    size_t n_div_4 = n / 4;

    #pragma unroll
    for(int el_idx = 0; el_idx < n_el_per_thread; el_idx++) {
        int idx = index + el_idx * stride;
        if(idx >= n_div_4)
            break;
        
        float4* Y_loc = &Y[idx];
        float4 Y_vec = *Y_loc;
        
        // float Y_prev = z_prev[idx];
        float4* z_prev_loc = &z_prev[idx];
        float4 z_prev_vec = *z_prev_loc;

        // float Y_val = max(0.0f, Y[idx] - alpha_L);
        float4 Y_val;
        Y_val.x = branchless_relu(Y_vec.x - alpha_L);
        Y_val.y = branchless_relu(Y_vec.y - alpha_L);
        Y_val.z = branchless_relu(Y_vec.z - alpha_L);
        Y_val.w = branchless_relu(Y_vec.w - alpha_L);

        // z_prev[idx] = Y_val;
        *z_prev_loc = Y_val;

        // float diff = Y_val - Y_prev;
        float4 diff;
        diff.x = Y_val.x - z_prev_vec.x;
        diff.y = Y_val.y - z_prev_vec.y;
        diff.z = Y_val.z - z_prev_vec.z;
        diff.w = Y_val.w - z_prev_vec.w;
        
        // thread_local_prev_z_norm += Y_prev * Y_prev;
        thread_local_prev_z_norm += z_prev_vec.x * z_prev_vec.x + z_prev_vec.y * z_prev_vec.y + z_prev_vec.z * z_prev_vec.z + z_prev_vec.w * z_prev_vec.w;

        // thread_local_diff_norm += diff * diff;
        thread_local_diff_norm += diff.x * diff.x + diff.y * diff.y + diff.z * diff.z + diff.w * diff.w;

        // Y_val += mlt * diff;
        Y_val.x += mlt * diff.x;
        Y_val.y += mlt * diff.y;
        Y_val.z += mlt * diff.z;
        Y_val.w += mlt * diff.w;

        // Y[idx] = Y_val;
        *Y_loc = Y_val;
    }

    {
        // tree-based reduction of thread-local norm values for all threads in the block
        // https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
        extern __shared__ float shmem[];
        float* shared_diff_norm = shmem;
        float* shared_prev_z_norm = &shmem[blockDim.x];

        shared_diff_norm[tid] = thread_local_diff_norm;
        shared_prev_z_norm[tid] = thread_local_prev_z_norm;
        __syncthreads();

        if(block_sz >= 512) {
            if(tid < 256) {
                shared_diff_norm[tid] += shared_diff_norm[tid + 256];
                shared_prev_z_norm[tid] += shared_prev_z_norm[tid + 256];
            }
            __syncthreads();
        }
        if(block_sz >= 256) {
            if(tid < 128) {
                shared_diff_norm[tid] += shared_diff_norm[tid + 128];
                shared_prev_z_norm[tid] += shared_prev_z_norm[tid + 128];
            }
            __syncthreads();
        }
        if(block_sz >= 128) {
            if(tid < 64) {
                shared_diff_norm[tid] += shared_diff_norm[tid + 64];
                shared_prev_z_norm[tid] += shared_prev_z_norm[tid + 64];
            }
            __syncthreads();
        }
        if(tid < 32) {
            warp_reduce<block_sz>(shared_diff_norm, tid);
            warp_reduce<block_sz>(shared_prev_z_norm, tid);
        }

        if (tid == 0) {
            atomicAdd(diff_norm, shared_diff_norm[0]);
            atomicAdd(prev_z_norm, shared_prev_z_norm[0]);
        }
    }
}

void print_norm_host(float* arr, size_t sz, const char* str) {
    double norm = 0;
    for(int idx = 0; idx < sz; idx++) {
        norm += (double)arr[idx] * (double)arr[idx];
    }
    norm = sqrt(norm);
    printf("%s: %f\n", str, norm);
}

void print_norm(float* arr_dev, size_t sz, const char* str) {
    float* arr = (float*)malloc(sz * sizeof(float));
    CHECK(arr);
    CHECK_CUDA_NORET(hipMemcpy((void*)arr, arr_dev, sz * sizeof(float), hipMemcpyDeviceToHost))
    print_norm_host(arr, sz, str);
    free(arr);
}

void cuda_log_time_diff(char* msg, hipEvent_t* start, hipEvent_t* stop) {
    float milli = 0;
    hipEventElapsedTime(&milli, *start, *stop);
    milli /= 1000;      // ms -> s
    printf("%s: %f\n", msg, milli);
}

void log_time_diff(char* msg, struct timeval* start, struct timeval* stop) {
    double start_ms = (((double)start->tv_sec)*1000)+(((double)start->tv_usec)/1000);
    double stop_ms = (((double)stop->tv_sec)*1000)+(((double)stop->tv_usec)/1000);
    double diff_in_sec = (stop_ms - start_ms)/1000;
    printf("%s: %f\n", msg, diff_in_sec);
}


extern "C" {
int fista(float* __restrict__ X_host, float* __restrict__ basis_host, float* __restrict__ Z_host, int n_samples, int inp_dim, int dict_sz, float lr, float alpha_L, int n_iter, float converge_thresh, int gpu_idx) {
    CHECK(X_host);
    CHECK(basis_host);
    CHECK(Z_host);

    struct timeval actual_start, handle_time, init, exec, result;
    if(DEBUG)
        gettimeofday(&actual_start, NULL);

    CHECK_CUDA_NORET(hipSetDevice(gpu_idx));

    // X: n_samples x inp_dim
    // basis: inp_dim x dict_sz
    // Z: n_samples x dict_sz

    // TODO(as): (very) slow the first time it is called in a process...
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    if(DEBUG)
        gettimeofday(&handle_time, NULL);

    // test_fista shows minor max difference (1e-4) from the reference solution when this option is enabled, but it doubles the performance of the BLAS section
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;

    // TODO(as): page locking + async transfers. ways to move these calls outside of this function so not called on each iteration?
    float *X, *basis;
    size_t x_n_el = n_samples * inp_dim;
    size_t x_sz = x_n_el * sizeof(float);
    size_t basis_sz = inp_dim * dict_sz * sizeof(float);
    CHECK_CUDA_NORET(allocateCompressible((void**)&X, x_sz))
    CHECK_CUDA_NORET(allocateCompressible((void**)&basis, basis_sz))
    CHECK_CUDA_NORET(hipMemcpy((void*)X, X_host, x_sz, hipMemcpyHostToDevice))
    CHECK_CUDA_NORET(hipMemcpy((void*)basis, basis_host, basis_sz, hipMemcpyHostToDevice))

    float *residual, *z_prev, *Y;
    size_t z_n_el = dict_sz * n_samples;
    size_t z_sz = z_n_el * sizeof(float);
    CHECK_CUDA_NORET(allocateCompressible((void**)&residual, x_sz))
    CHECK_CUDA_NORET(allocateCompressible((void**)&z_prev, z_sz))
    CHECK_CUDA_NORET(allocateCompressible((void**)&Y, z_sz))
    CHECK_CUDA_NORET(hipMemset(z_prev, 0, z_sz))
    CHECK_CUDA_NORET(hipMemset(Y, 0, z_sz))


    CHECK(z_n_el % 4 == 0);         // assumed by kernel format

    float norms_host[2];
    float* norms;
    size_t norm_sz = 2 * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&norms, norm_sz))

    if(DEBUG)
        gettimeofday(&init, NULL);

    float tk = 1, tk_prev = 1;
    int itr;
    for(itr = 0; itr < n_iter; itr++) {

        hipEvent_t start, blas, k_start, k_exec, k_end;
        if(DEBUG) {    
            hipEventCreate(&start);
            hipEventCreate(&blas);
            hipEventCreate(&k_start);
            hipEventCreate(&k_exec);
            hipEventCreate(&k_end);
            hipEventRecord(start);
        }

        // residual = x - (z @ basis.T)
        CHECK_CUDA_NORET(hipMemcpy((void*)residual, X, x_sz, hipMemcpyDeviceToDevice))
        {
            // cublas assumes column-major but we have row major
            // https://i.sstatic.net/IvZPe.png
            float alpha = -1.0f;
            float beta = 1.0f;
            CHECK_CUBLAS_NORET(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inp_dim, n_samples, dict_sz, &alpha, basis, HIP_R_32F, dict_sz, Y, HIP_R_32F, dict_sz, &beta, residual, HIP_R_32F, inp_dim, compute_type, HIPBLAS_GEMM_DEFAULT));    
        }

        // mm = residual @ basis
        // z += lr * mm
        {
            // cublas assumes column-major but we have row major
            // https://i.sstatic.net/IvZPe.png
            float beta = 1.0f;
            CHECK_CUBLAS_NORET(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dict_sz, n_samples, inp_dim, &lr, basis, HIP_R_32F, dict_sz, residual, HIP_R_32F, inp_dim, &beta, Y, HIP_R_32F, dict_sz, compute_type, HIPBLAS_GEMM_DEFAULT));
        }

        if(DEBUG) hipEventRecord(blas);

        // Y-update multiplier
        tk_prev = tk;
        tk = (1 + sqrtf(1 + 4 * tk * tk)) / 2;
        float mlt = (tk_prev - 1) / tk;
        CHECK_CUDA_NORET(hipMemset(norms, 0, norm_sz))
        if(DEBUG) hipEventRecord(k_start);
        
        const int block_sz = 384;
        const int n_el_per_thread = 16;
        int n_blocks = (int)ceil((float)z_n_el / (float)(n_el_per_thread * block_sz));
        if(DEBUG) printf("nblocks: %d\n", n_blocks);
        int smem_sz = 2 * block_sz * sizeof(float);
        y_update<block_sz, n_el_per_thread><<<n_blocks, block_sz, smem_sz>>>(z_n_el, (float4*)Y, (float4*)z_prev, alpha_L, mlt, norms, &norms[1]);
        if(DEBUG) hipEventRecord(k_exec);

        CHECK_CUDA_NORET(hipMemcpy((void*)norms_host, norms, norm_sz, hipMemcpyDeviceToHost))

        // Frobenius norm can be defined as the L2 norm of the flattened matrix
        float diff_norm = norms_host[0];
        float prev_z_norm = norms_host[1];
        float norm_ratio = diff_norm / prev_z_norm;
        norm_ratio = sqrtf(norm_ratio);         // equivalent to sqrtf(diff_norm) / sqrtf(prev_z_norm)
        if(DEBUG) hipEventRecord(k_end);


        if(DEBUG) {
            printf("\33[2K\r%d / %d", itr, n_iter);
            fflush(stdout);

            hipEventSynchronize(k_end);
            cuda_log_time_diff("\n\tblas", &start, &blas);
            cuda_log_time_diff("\tk_start", &blas, &k_start);
            cuda_log_time_diff("\tk_exec", &k_start, &k_exec);
            cuda_log_time_diff("\tk_end", &k_exec, &k_end);
            float milli = 0;
            hipEventElapsedTime(&milli, k_start, k_exec);
            printf("\tbandwidth: %f (GB/s)\n", z_sz * 4 / milli / 1e6);     // 2 read + 2 write per iteration   

            hipEventDestroy(start);
            hipEventDestroy(blas);
            hipEventDestroy(k_start);
            hipEventDestroy(k_exec);
            hipEventDestroy(k_end);
        }

        if(itr != 0 && norm_ratio < converge_thresh)
            break;
    }
    if(DEBUG) {
        printf("\n");
        gettimeofday(&exec, NULL);
    }

    // memcpy(Z, z_prev, dict_sz * n_samples * sizeof(float));
    CHECK_CUDA_NORET(hipMemcpy((void*)Z_host, z_prev, z_sz, hipMemcpyDeviceToHost))


    CHECK_CUDA_NORET(freeCompressible(residual, x_sz))
    CHECK_CUDA_NORET(freeCompressible(z_prev, z_sz))
    CHECK_CUDA_NORET(hipFree(norms))
    CHECK_CUDA_NORET(freeCompressible(Y, z_sz))
    CHECK_CUDA_NORET(freeCompressible(X, x_sz))
    CHECK_CUDA_NORET(freeCompressible(basis, basis_sz))

    hipblasDestroy(handle);

    if(DEBUG) {
        gettimeofday(&result, NULL);
        log_time_diff("\n\n\thandle", &actual_start, &handle_time);
        log_time_diff("\tinit", &handle_time, &init);
        log_time_diff("\texec", &init, &exec);
        log_time_diff("\tresult", &exec, &result);
    }
    return itr;
}
}


void print_stack_trace() {
    void *array[100];
    size_t size;
    char **strings;
    size_t i;

    size = backtrace(array, 100);
    strings = backtrace_symbols(array, size);

    printf("Stack trace:\n");
    for (i = 0; i < size; i++) {
        printf("\t%s\n", strings[i]);
    }

    free(strings);
}


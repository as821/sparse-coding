#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <execinfo.h>
#include <sys/time.h>
#include <stdbool.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>


void print_stack_trace();
#define CHECK(x)                                                                                    \
{                                                                                                   \
    if(!(x)) {                                                                                      \
        printf("ERROR (line %d, file:%s) (%d): %s\n", __LINE__, __FILE__, errno, strerror(errno));  \
        print_stack_trace();                                                                        \
        exit(EXIT_FAILURE);                                                                         \
    }                                                                                               \
}

#define CHECK_CUDA_NORET(func)                                                  \
{                                                                               \
    hipError_t status = (func);                                                \
    if (status != hipSuccess) {                                                \
        printf("CUDA API failed at line %d with error: %s (%d) (%s)\n",         \
               __LINE__, hipGetErrorString(status), status, __FILE__);         \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define CHECK_CUBLAS_NORET(func)                                                \
{                                                                               \
    hipblasStatus_t status = (func);                                             \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                      \
        printf("CUBLAS API failed at line %d with error: (%d) (%s)\n",          \
               __LINE__, status, __FILE__);                                     \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}


__device__ __forceinline__ float branchless_relu(float x) {
    return x * (x > 0.0f);
}

template <unsigned int block_sz>
__device__ __forceinline__ void warp_reduce(volatile float* sdata, int tid) {
    if(block_sz >= 64)
        sdata[tid] += sdata[tid + 32];
    if(block_sz >= 32)
        sdata[tid] += sdata[tid + 16];
    if(block_sz >= 16)
        sdata[tid] += sdata[tid + 8];
    if(block_sz >= 8)
        sdata[tid] += sdata[tid + 4];
    if(block_sz >= 4)
        sdata[tid] += sdata[tid + 2];
    if(block_sz >= 2)
        sdata[tid] += sdata[tid + 1];
}

template <unsigned int block_sz, unsigned int n_el_per_thread>
__global__ void y_update(size_t n, float4* __restrict__ Y, float4* __restrict__ z_prev, float alpha_L, float* _mlt, float* __restrict__ diff_norm, float* __restrict__ prev_z_norm) {
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;
    
    float thread_local_diff_norm = 0;
    float thread_local_prev_z_norm = 0;

    size_t n_div_4 = n / 4;

    #pragma unroll
    for(int el_idx = 0; el_idx < n_el_per_thread; el_idx++) {
        int idx = index + el_idx * stride;
        if(idx >= n_div_4)
            break;
        
        float4* Y_loc = &Y[idx];
        float4 Y_vec = *Y_loc;
        
        // float Y_prev = z_prev[idx];
        float4* z_prev_loc = &z_prev[idx];
        float4 z_prev_vec = *z_prev_loc;

        // float Y_val = max(0.0f, Y[idx] - alpha_L);
        float4 Y_val;
        Y_val.x = branchless_relu(Y_vec.x - alpha_L);
        Y_val.y = branchless_relu(Y_vec.y - alpha_L);
        Y_val.z = branchless_relu(Y_vec.z - alpha_L);
        Y_val.w = branchless_relu(Y_vec.w - alpha_L);

        // z_prev[idx] = Y_val;
        *z_prev_loc = Y_val;

        // float diff = Y_val - Y_prev;
        float4 diff;
        diff.x = Y_val.x - z_prev_vec.x;
        diff.y = Y_val.y - z_prev_vec.y;
        diff.z = Y_val.z - z_prev_vec.z;
        diff.w = Y_val.w - z_prev_vec.w;
        
        // thread_local_prev_z_norm += Y_prev * Y_prev;
        thread_local_prev_z_norm += z_prev_vec.x * z_prev_vec.x + z_prev_vec.y * z_prev_vec.y + z_prev_vec.z * z_prev_vec.z + z_prev_vec.w * z_prev_vec.w;

        // thread_local_diff_norm += diff * diff;
        thread_local_diff_norm += diff.x * diff.x + diff.y * diff.y + diff.z * diff.z + diff.w * diff.w;

        // Y_val += mlt * diff;
        float mlt = *_mlt;
        Y_val.x += mlt * diff.x;
        Y_val.y += mlt * diff.y;
        Y_val.z += mlt * diff.z;
        Y_val.w += mlt * diff.w;

        // Y[idx] = Y_val;
        *Y_loc = Y_val;
    }

    {
        // tree-based reduction of thread-local norm values for all threads in the block
        // https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
        extern __shared__ float shmem[];
        float* shared_diff_norm = shmem;
        float* shared_prev_z_norm = &shmem[blockDim.x];

        shared_diff_norm[tid] = thread_local_diff_norm;
        shared_prev_z_norm[tid] = thread_local_prev_z_norm;
        __syncthreads();

        if(block_sz >= 512) {
            if(tid < 256) {
                shared_diff_norm[tid] += shared_diff_norm[tid + 256];
                shared_prev_z_norm[tid] += shared_prev_z_norm[tid + 256];
            }
            __syncthreads();
        }
        if(block_sz >= 256) {
            if(tid < 128) {
                shared_diff_norm[tid] += shared_diff_norm[tid + 128];
                shared_prev_z_norm[tid] += shared_prev_z_norm[tid + 128];
            }
            __syncthreads();
        }
        if(block_sz >= 128) {
            if(tid < 64) {
                shared_diff_norm[tid] += shared_diff_norm[tid + 64];
                shared_prev_z_norm[tid] += shared_prev_z_norm[tid + 64];
            }
            __syncthreads();
        }
        if(tid < 32) {
            warp_reduce<block_sz>(shared_diff_norm, tid);
            warp_reduce<block_sz>(shared_prev_z_norm, tid);
        }

        if (tid == 0) {
            atomicAdd(diff_norm, shared_diff_norm[0]);
            atomicAdd(prev_z_norm, shared_prev_z_norm[0]);
        }
    }
}

void print_norm_host(float* arr, size_t sz, const char* str) {
    double norm = 0;
    for(int idx = 0; idx < sz; idx++) {
        norm += (double)arr[idx] * (double)arr[idx];
    }
    norm = sqrt(norm);
    printf("%s: %f\n", str, norm);
}

void print_norm(float* arr_dev, size_t sz, const char* str) {
    float* arr = (float*)malloc(sz * sizeof(float));
    CHECK(arr);
    CHECK_CUDA_NORET(hipMemcpy((void*)arr, arr_dev, sz * sizeof(float), hipMemcpyDeviceToHost))
    print_norm_host(arr, sz, str);
    free(arr);
}

void cuda_log_time_diff(char* msg, hipEvent_t* start, hipEvent_t* stop) {
    float milli = 0;
    hipEventElapsedTime(&milli, *start, *stop);
    milli /= 1000;      // ms -> s
    printf("%s: %f\n", msg, milli);
}

void log_time_diff(char* msg, struct timeval* start, struct timeval* stop) {
    double start_ms = (((double)start->tv_sec)*1000)+(((double)start->tv_usec)/1000);
    double stop_ms = (((double)stop->tv_sec)*1000)+(((double)stop->tv_usec)/1000);
    double diff_in_sec = (stop_ms - start_ms)/1000;
    printf("%s: %f\n", msg, diff_in_sec);
}


extern "C" {
int fista(float* __restrict__ X_host, float* __restrict__ basis_host, float* __restrict__ Z_host, int n_samples, int inp_dim, int dict_sz, float lr, float alpha_L, int n_iter, float converge_thresh) {
    CHECK(X_host);
    CHECK(basis_host);
    CHECK(Z_host);

    struct timeval actual_start, handle_time, init, exec;
    gettimeofday(&actual_start, NULL);


    // X: n_samples x inp_dim
    // basis: inp_dim x dict_sz
    // Z: n_samples x dict_sz

    // TODO(as): (very) slow the first time it is called in a process...
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    gettimeofday(&handle_time, NULL);

    // test_fista shows minor max difference (1e-4) from the reference solution when this option is enabled, but it doubles the performance of the BLAS section
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;

    // TODO(as): page locking + async transfers. ways to move these calls outside of this function so not called on each iteration?
    float *X, *basis;
    size_t x_n_el = n_samples * inp_dim;
    size_t x_sz = x_n_el * sizeof(float);
    size_t basis_sz = inp_dim * dict_sz * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&X, x_sz))
    CHECK_CUDA_NORET(hipMalloc((void**)&basis, basis_sz))
    CHECK_CUDA_NORET(hipMemcpy((void*)X, X_host, x_sz, hipMemcpyHostToDevice))
    CHECK_CUDA_NORET(hipMemcpy((void*)basis, basis_host, basis_sz, hipMemcpyHostToDevice))

    float *residual, *z_prev, *Y;
    size_t z_n_el = dict_sz * n_samples;
    size_t z_sz = z_n_el * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&residual, x_sz))
    CHECK_CUDA_NORET(hipMalloc((void**)&z_prev, z_sz))
    CHECK_CUDA_NORET(hipMalloc((void**)&Y, z_sz))
    CHECK_CUDA_NORET(hipMemset(z_prev, 0, z_sz))
    CHECK_CUDA_NORET(hipMemset(Y, 0, z_sz))


    CHECK(z_n_el % 4 == 0);         // assumed by kernel format

    float norms_host[2];
    float* norms;
    size_t norm_sz = 2 * sizeof(float);
    CHECK_CUDA_NORET(hipMalloc((void**)&norms, norm_sz))

    float *d_mlt;
    CHECK_CUDA_NORET(hipMalloc(&d_mlt, sizeof(float)));

    gettimeofday(&init, NULL);

    float tk = 1, tk_prev = 1;

    // TODO(as) graph capture

    hipStream_t stream;
    hipStreamCreate(&stream);
    CHECK_CUBLAS_NORET(hipblasSetStream(handle, stream));
    // CHECK_CUBLAS_NORET(cublasSetPointerMode(handle, CUBLAS_POINTER_MODE_DEVICE));

    // cudaEvent_t start, blas, k_start, k_exec, k_end;
    // cudaEventCreate(&start);
    // cudaEventCreate(&blas);
    // cudaEventCreate(&k_start);
    // cudaEventCreate(&k_exec);
    // cudaEventCreate(&k_end);

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // residual = x - (z @ basis.T)
    CHECK_CUDA_NORET(hipMemcpyAsync((void*)residual, X, x_sz, hipMemcpyDeviceToDevice, stream))
    {
        // cublas assumes column-major but we have row major
        // https://i.sstatic.net/IvZPe.png
        float alpha = -1.0f;
        float beta = 1.0f;
        CHECK_CUBLAS_NORET(hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, inp_dim, n_samples, dict_sz, &alpha, basis, HIP_R_32F, dict_sz, Y, HIP_R_32F, dict_sz, &beta, residual, HIP_R_32F, inp_dim, compute_type, HIPBLAS_GEMM_DEFAULT));    
    }

    // mm = residual @ basis
    // z += lr * mm
    {
        // cublas assumes column-major but we have row major
        // https://i.sstatic.net/IvZPe.png
        float beta = 1.0f;
        CHECK_CUBLAS_NORET(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dict_sz, n_samples, inp_dim, &lr, basis, HIP_R_32F, dict_sz, residual, HIP_R_32F, inp_dim, &beta, Y, HIP_R_32F, dict_sz, compute_type, HIPBLAS_GEMM_DEFAULT));
    }

    CHECK_CUDA_NORET(hipMemsetAsync(norms, 0, norm_sz, stream))
    
    const int block_sz = 384;
    const int n_el_per_thread = 16;
    int n_blocks = (int)ceil((float)z_n_el / (float)(n_el_per_thread * block_sz));
    printf("nblocks: %d\n", n_blocks);
    int smem_sz = 2 * block_sz * sizeof(float);
    y_update<block_sz, n_el_per_thread><<<n_blocks, block_sz, smem_sz, stream>>>(z_n_el, (float4*)Y, (float4*)z_prev, alpha_L, d_mlt, norms, &norms[1]);

    hipGraph_t graph;
    hipGraphExec_t instance;
    CHECK_CUDA_NORET(hipStreamEndCapture(stream, &graph));
    CHECK_CUDA_NORET(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));





    int itr;
    for(itr = 0; itr < n_iter; itr++) {

        

        // Y-update multiplier
        tk_prev = tk;
        tk = (1 + sqrtf(1 + 4 * tk * tk)) / 2;
        float mlt = (tk_prev - 1) / tk;
        CHECK_CUDA_NORET(hipMemcpyAsync(d_mlt, &mlt, sizeof(float), hipMemcpyHostToDevice, stream));


        // TODO(as) graph replay
        CHECK_CUDA_NORET(hipGraphLaunch(instance, stream));
        CHECK_CUDA_NORET(hipStreamSynchronize(stream));



        CHECK_CUDA_NORET(hipMemcpy((void*)norms_host, norms, norm_sz, hipMemcpyDeviceToHost))

        // Frobenius norm can be defined as the L2 norm of the flattened matrix
        float diff_norm = norms_host[0];
        float prev_z_norm = norms_host[1];
        float norm_ratio = diff_norm / prev_z_norm;
        norm_ratio = sqrtf(norm_ratio);         // equivalent to sqrtf(diff_norm) / sqrtf(prev_z_norm)

        // printf("%d: %f %f\n", itr, sqrtf(diff_norm), sqrtf(prev_z_norm));
        
        printf("\33[2K\r%d / %d", itr, n_iter);
        fflush(stdout);

        // cuda_log_time_diff("\n\tblas", &start, &blas);
        // cuda_log_time_diff("\tk_start", &blas, &k_start);
        // cuda_log_time_diff("\tk_exec", &k_start, &k_exec);
        // cuda_log_time_diff("\tk_end", &k_exec, &k_end);
        // float milli = 0;
        // cudaEventElapsedTime(&milli, k_start, k_exec);
        // printf("\tbandwidth: %f (GB/s)\n", z_sz * 4 / milli / 1e6);     // 2 read + 2 write per iteration   

        if(itr != 0 && norm_ratio < converge_thresh)
            break;
    }
    // printf("\n");

    CHECK_CUDA_NORET(hipGraphExecDestroy(instance));
    CHECK_CUDA_NORET(hipGraphDestroy(graph));
    CHECK_CUDA_NORET(hipStreamDestroy(stream));

    // memcpy(Z, z_prev, dict_sz * n_samples * sizeof(float));
    CHECK_CUDA_NORET(hipMemcpy((void*)Z_host, z_prev, z_sz, hipMemcpyDeviceToHost))


    CHECK_CUDA_NORET(hipFree(residual))
    CHECK_CUDA_NORET(hipFree(z_prev))
    CHECK_CUDA_NORET(hipFree(norms))
    CHECK_CUDA_NORET(hipFree(Y))
    CHECK_CUDA_NORET(hipFree(X))
    CHECK_CUDA_NORET(hipFree(basis))

    hipblasDestroy(handle);


    gettimeofday(&exec, NULL);


    log_time_diff("\n\n\thandle", &actual_start, &handle_time);
    log_time_diff("\tinit", &handle_time, &init);
    log_time_diff("\texec", &init, &exec);
    return itr;
}
}


void print_stack_trace() {
    void *array[100];
    size_t size;
    char **strings;
    size_t i;

    size = backtrace(array, 100);
    strings = backtrace_symbols(array, size);

    printf("Stack trace:\n");
    for (i = 0; i < size; i++) {
        printf("\t%s\n", strings[i]);
    }

    free(strings);
}

